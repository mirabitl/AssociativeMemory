#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples
#include <hip/device_functions.h> // helper functions for SDK examples



#include "combiStruct.h"

void allocateLayers()
{
  checkCudaErrors(hipMalloc((void **) &d_layer,GPU_MAX_LAYER *sizeof(combiLayer)));
  checkCudaErrors(hipMalloc((void **) &d_cand,GPU_MAX_CAND *sizeof(combiTrack)));
  checkCudaErrors(hipHostAlloc((void **)&h_layer,GPU_MAX_LAYER *sizeof(combiLayer),hipHostMallocWriteCombined));
  checkCudaErrors(hipHostAlloc((void **)&h_cand,GPU_MAX_CAND *sizeof(combiTrack),hipHostMallocWriteCombined));

}

void freeLayers()
{
  checkCudaErrors(hipHostFree(h_layer));
  checkCudaErrors(hipHostFree(h_cand));
  checkCudaErrors(hipFree(d_layer));
  checkCudaErrors(hipFree(d_cand));
}

void clearLayers()
{
  memset(h_layers,0,GPU_MAX_LAYER *sizeof(combiLayer));
  checkCudaErrors(hipMemset(d_layer,0,GPU_MAX_LAYER *sizeof(combiLayer)));
}
void clearCandidates()
{
  memset(h_cand,0,GPU_MAX_CAND *sizeof(combiTrack));
  checkCudaErrors(hipMemset(d_cand,0,GPU_MAX_CAND *sizeof(combiTrack)));
}

void copyLayer(uint32_t idl)
{
  checkCudaErrors(hipMemcpy(&d_layers[idl], &h_layer[idl],
			     sizeof(int)+h_layer[idl]*sizeof(stubPosition),hipMemcpyHostToDevice));
}

__global__ void
computeLayerKernel(combiLayer* L)
{
  const unsigned int ib=threadIdx.x;
  L->stub[ib]._r2= L->stub[ib]._x*L->stub[ib]._x+L->stub[ib]._y*L->stub[ib]._y;
  L->stub[ib]._r=sqrt(L->stub[ib]._r2);
  L->stub[ib]._xp=L->stub[ib]._x/L->stub[ib]._r2;
  L->stub[ib]._yp=L->stub[ib]._y/L->stub[ib]._r2;
  
}

void computeLayer(uint32_t idl)
{
  computeLayerKernel<<<1,h_layer[idl]._nb,0>>>(&d_layer[idl]);
}
